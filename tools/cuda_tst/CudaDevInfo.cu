/** 
    \file
*/

#include "CudaDevInfo.h"

#define REPORT_CUDA_INIT 0	/**< If true, report when initializing */
#define RESET_CUDA_INIT 0	/**< If true, reset device when initializing */

int GPUInfo::getDeviceIndex()
{
  int indx;
 
  gpuErrChk(hipGetDevice(&indx), "GPUInfo:getDeviceIndex:error", "Unexpected error");

  return indx;
}

int GPUInfo::deviceCount()
{
  int ndev;
 
  gpuErrChk(hipGetDeviceCount(&ndev), "GPUInfo:deviceCount:error", "Unexpected error");

  return ndev;
}

void GPUInfo::init(int dev_ind)
{
  gpuErrChk(hipSetDevice(dev_ind),
	     "GPUInfo:init:hipSetDevice","Error setting GPU device");
  dev_indx = getDeviceIndex();
	     
#if RESET_CUDA_INIT
  gpuErrChk(hipDeviceReset(),"","");
#endif

#if REPORT_CUDA_INIT
  printf("Initializing CUDA cuda_dev=0x%lX\n", (unsigned long)cuda_dev);
  printf("Device Index: %d\n", cuda_dev->dev_indx);
#endif

  gpuErrChk(hipGetDeviceProperties(&prop, dev_indx),
	     "GPUInfo:init:hipGetDeviceProperties","Error getting GPU properties");

  // Compute log2_thrds_blk
  log2_thrds_blk = 0;
  for(int m=prop.maxThreadsPerBlock; m>1; m >>= 1)
    log2_thrds_blk++;

  // Force some action
  void *p;
  const size_t len = 10;
  gpuErrChk(hipMalloc(&p , len), "", "Alloc error");
  gpuErrChk(hipMemset(p, 0, len), "", "memset error");
  gpuErrChk(hipFree(p), "", "Alloc error");

#ifdef MATLAB_MEX_FILES

  gpuErrChk((mxInitGPU() == MX_GPU_SUCCESS)? hipSuccess: hipErrorNotInitialized,
	     ("GPUInfo:init:mxInitGPU", "mxInitGPU failed"));

#endif
}

GlobalPtr<GPUInfo> cuda_dev("cuda_dev");

