#include "hip/hip_runtime.h"
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <stdio.h>
#include <errno.h>
#include <unistd.h>
#include <stdlib.h>
#include <arpa/inet.h>
#include <math.h>
#include "cs_cuda.h"
#include "cs_dbg.h"
#include "cs_helper.h"
#include "cs_copy_box.h"

// #define CUDA_DBG
// #define CUDA_DBG1

// ex, ey, and exy_size are the embedded dimensions/size
// ex/ey are from the point to the edge of the cx/cy

template<typename T>
__global__ void d_do_copy_vec ( T *fdp, T *tdp, int tbl_size, 
	int from_size, int to_size )
{
	int t_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int k ;

	while ( t_idx < tbl_size )
	{
		k = ( t_idx / to_size ) * from_size + ( t_idx % to_size ) ;

		tdp[ t_idx ] = fdp[ k ] ;

		t_idx += CUDA_MAX_THREADS ;
	}		
}


template<typename T> int
h_do_copy_vec ( T *fromp, T *top, int total_size, int from_size,
	int to_size ) 
{
	int nThreadsPerBlock = CUDA_MAX_THREADS_P_BLK ;
	int nBlocks ; // = ( total_size + ( nThreadsPerBlock - 1 ))/nThreadsPerBlock ;

#ifdef CUDA_OBS 
	printf("%s ...\n", __func__ ) ;
	fprintf(stderr, "%s: f %p t %p total %d from %d to %d \n",
		__func__, fromp, top, total_size, from_size, to_size ) ; 
#endif

	if (( total_size % to_size ) || ( to_size > from_size ))
	{
		fprintf( stderr, "h_do_copy_vec: size %d %d %d\n", total_size, from_size, to_size ) ;
		return ( 0 ) ;
	} 

#ifdef CUDA_OBS 
	dbg_p_d_data_f("copy_vec before ", ( float *)fromp, 900 ) ; 
#endif 

	h_block_adj ( total_size, nThreadsPerBlock, &nBlocks ) ;

	d_do_copy_vec<T> <<< nBlocks, nThreadsPerBlock >>> (
		fromp, top, total_size, from_size, to_size ) ;

	hipDeviceSynchronize() ;

#ifdef CUDA_OBS 
	dbg_p_d_data_f("copy_vec", ( float *)top, total_size ) ; 
#endif 
	return ( 1 ) ;
}

// ex, ey, and exy_size are the embedded dimensions/size
// ex/ey are from the point to the edge of the cx/cy
// obxyz_size is the old inner block size
// nbxyz_size is the new inner block size 
// tbl_size is the old size

template<typename T>
__global__ void d_do_copy_box_v2 ( T *fdp, T *tdp, int tbl_size, 
	int ex, int ey, int obxyz_size, int nbxyz_size, struct cube *d_xyzp,
	int blk_in_x, int blk_in_y )
{
	int f_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int cx, cy, cz, xy_size, exy_size, blk_idx, frame, block, i, j, x, y ;
	T *otdp ;

	otdp = tdp ;
	while ( f_idx < tbl_size )
	{
		tdp = otdp ;

		block = f_idx / obxyz_size ; // which block

		i = block % blk_in_x ; 	// 0..blk_in_x-1
		j = block / blk_in_x ;	// 0..blk_in_y-1

		if (( i == 0 ) || ( i == ( blk_in_x - 1 ))) 
		{
			if (( j == 0 ) || ( j == ( blk_in_y - 1 )))
				blk_idx = 2 ;
			else
				blk_idx = 1 ;
		} else
		{
			if (( j == 0 ) || ( j == ( blk_in_y - 1 )))
				blk_idx = 1 ;
			else
				blk_idx = 0 ;
		}

		i = f_idx % obxyz_size ;	// mea offset in the block

		cx = d_xyzp[ blk_idx ].x ;
		cy = d_xyzp[ blk_idx ].y ;
		cz = d_xyzp[ blk_idx ].z ;

		xy_size = cx * cy ;
		exy_size = ( cx - ex * 2 ) * ( cy - ey * 2 ) ;

		frame = i / xy_size ;
		
		i = i % xy_size ;
		y = i / cx ;
		x = i % cx ;

		if (( frame < cz ) && ( y >= ey ) && ( x >= ex ) && 
			(( cy - y ) > ey ) && (( cx - x ) > ex )) 
		{
			tdp += block * nbxyz_size + frame * exy_size ;
			i = ( y - ey ) * ( cx - 2 * ex ) + ( x - ex ) ;
			
			tdp [ i ] = fdp [ f_idx ] ;
		} 

		f_idx += CUDA_MAX_THREADS ;

	}		
}

template<typename T>
int
h_do_copy_box_v2 ( T *fromp, T *top, int tbl_size, 
	int edge_x, int edge_y, int blk_in_x, int blk_in_y, struct cube *d_cp, 
	struct cube *cp )
{
	int nThreadsPerBlock = CUDA_MAX_THREADS_P_BLK ;
	int nBlocks ; // = ( tbl_size + ( nThreadsPerBlock - 1 ))/nThreadsPerBlock ;
	int obxyz_size = cp[0].x * cp[0].y * cp[0].z ;
	int nbxyz_size = (( cp[0].x - ( edge_x * 2 )) *
		(( cp[0].y - ( edge_y * 2 ))) * cp[0].z ) ;
#ifdef CUDA_DBG 
	int i ;
#endif 
	struct cube temp_cube[ CUBE_INFO_CNT ] ;

	memcpy ( &temp_cube, cp, sizeof ( *cp )) ;

#ifdef CUDA_DBG1 
	fprintf(stderr, "%s: f %p t %p tblsize %d edge %d %d blk %d %d cubep %p\n",
		__func__, fromp, top, tbl_size, edge_x, edge_y, blk_in_x,
		blk_in_y, cp ) ;
	fprintf(stderr, " 	nsize %d osize %d\n", nbxyz_size, obxyz_size ) ;
#endif 

	if ( tbl_size % obxyz_size )
	{
		fprintf(stderr, "%s: error size %d cube %d \n", __func__,
			tbl_size, obxyz_size ) ;
		return ( 0 ) ;
	}

	// the smallest cube ... 
	if ((( cp[2].x - ( edge_x * 2 )) < 0 ) ||
		(( cp[2].y - ( edge_y * 2 )) < 0 ))
	{
		fprintf(stderr, "%s: error cube %d %d edge %d %d\n",
			__func__, cp[2].x, cp[2].y, edge_x, edge_y ) ;
		return ( 0 ) ;
	}

 	h_block_adj ( tbl_size, nThreadsPerBlock, &nBlocks ) ;

	d_do_copy_box_v2<T> <<< nBlocks, nThreadsPerBlock >>> (
		fromp, top, tbl_size, edge_x, edge_y, obxyz_size, nbxyz_size,
		d_cp, blk_in_x, blk_in_y ) ;

	hipDeviceSynchronize() ;

#ifdef CUDA_OBS 

	for ( i = 0 ; i < CUBE_INFO_CNT ; i++ )
	{
		temp_cube[i].x -= edge_x * 2 ;
		temp_cube[i].y -= edge_y * 2 ;
	}

	dbg_p_d_data_i_mn_v2("copy_box_v2 done", top, ( tbl_size / obxyz_size ) *
		nbxyz_size, 100, temp_cube, blk_in_x, blk_in_y ) ;
#endif 

	return ( 1 ) ;
}

__global__ void d_do_copy_box ( int *fdp, int *tdp, int tbl_size, 
	int cx, int cy, int ex, int ey, int xy_size, int exy_size )
{
	int *otdp, f_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int block, i, x, y ;

	otdp = tdp ;
	while ( f_idx < tbl_size )
	{
		tdp = otdp ;

		block = f_idx / xy_size ;

		tdp += block * exy_size ;
		
		i = f_idx % xy_size ;
		y = i / cx ;
		x = i % cx ;

		if (( y >= ey ) && ( x >= ex ) && (( cy - y ) > ey ) && (( cx -x ) > ex )) 
		{
			i = ( y - ey ) * ( cx - 2 * ex ) + ( x - ex ) ;
			
			tdp [ i ] = fdp [ f_idx ] ;
		} 

		f_idx += CUDA_MAX_THREADS ;
	}		
}

// edge_x/y are the distance from the cube_x/y to the embedded box ( defined
// by edge_x/y )
int
h_do_copy_box ( int *fromp, int *top, int tbl_size, int cube_x,
	int cube_y, int edge_x, int edge_y )
{
	int nThreadsPerBlock = CUDA_MAX_THREADS_P_BLK ;
	int nBlocks ; // = ( tbl_size + ( nThreadsPerBlock - 1 ))/nThreadsPerBlock ;
	int cube_xy = cube_y * cube_x ;
	int edge_xy = (( cube_x - ( edge_x * 2 )) *
		(( cube_y - ( edge_y * 2 )))) ;

#ifdef CUDA_DBG1 
	fprintf(stderr, "%s: f %p t %p tblsize %d cube %d %d edge %d %d csize %d"
		" esize %d\n",
		__func__, fromp, top, tbl_size, cube_x, cube_y, edge_x, edge_y,
		cube_xy, edge_xy ) ;
#endif 

	if ( tbl_size % cube_xy )
	{
		fprintf(stderr, "%s: error size %d cube %d \n", __func__,
			tbl_size, cube_xy ) ;
		return ( 0 ) ;
	}

	if ((( cube_x - ( edge_x * 2 )) < 0 ) ||
		(( cube_y - ( edge_y * 2 )) < 0 ))
	{
		fprintf(stderr, "%s: error cube %d %d edge %d %d\n",
			__func__, cube_x, cube_y, edge_x, edge_y ) ;
		return ( 0 ) ;
	}

	h_block_adj ( tbl_size, nThreadsPerBlock, &nBlocks ) ;

	d_do_copy_box <<< nBlocks, nThreadsPerBlock >>> (
		fromp, top, tbl_size, cube_x, cube_y, edge_x, edge_y,
		cube_xy, edge_xy ) ;

	hipDeviceSynchronize() ;

#ifdef CUDA_OBS 
	dbg_p_d_data_i("copy_box", top, ( tbl_size / cube_xy ) * edge_xy ) ; 
#endif 
	return ( 1 ) ;
}

/*
   this routine copy the cubes in the all the blocks into the vector
   pointed by top.  from_size is the block size, to_size is the cube size
	total_size is the size of the copy ... in element
	copy the first from_size elements from fromp to top for every block
*/

template int
h_do_copy_vec<int> ( int *fromp, int *top, int total_size, int from_size,
	int to_size ) ;

template int
h_do_copy_vec<float> ( float *fromp, float *top, int total_size, int from_size,
	int to_size ) ;


template int
h_do_copy_box_v2<float> ( float *fromp, float *top, int tbl_size, 
	int edge_x, int edge_y, int blk_in_x, int blk_in_y, struct cube *d_cp, 
	struct cube *cp ) ;

template int
h_do_copy_box_v2<int> ( int *fromp, int *top, int tbl_size, 
	int edge_x, int edge_y, int blk_in_x, int blk_in_y, struct cube *d_cp, 
	struct cube *cp ) ;
