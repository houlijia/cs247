#include "hip/hip_runtime.h"
#include <json/json.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>

#include "json.h"
#include "cs_block.h"
#include "cs_config.h"
#include "cs_analysis.h"

#define CUDA_DBG 

void cs_config_getvalue( struct cs_config *csp, char *key, void *val ) ;

/*printing the value corresponding to boolean, double, integer and strings*/
void print_json_value
(json_object *jobj)
{
	enum json_type type;
	type = json_object_get_type(jobj); /*Getting the type of the json object*/
	// printf("type: ",type);
	switch (type) {
		case json_type_boolean: printf("json_type_boolean\n");
			printf("value: %s\n", json_object_get_boolean(jobj)? "true": "false");
			break;
		case json_type_double: printf("json_type_double\n");
			printf("          value: %lf\n", json_object_get_double(jobj));
			break;
		case json_type_int: printf("json_type_int\n");
			printf("          value: %d\n", json_object_get_int(jobj));
			break;
		case json_type_string: printf("json_type_string\n");
			printf("          value: %s\n", json_object_get_string(jobj));
			break;
	default:
	  assert(0);		// json_type_null or any other value should not happen
	}

}

void 
json_parse_array( struct cs_config *csp, json_object *jobj, char *key) 
{
	int json_parse( struct cs_config *csp, json_object * jobj); /*Forward Declaration*/
	enum json_type type;

	json_object *jarray = jobj; /*Simply get the array*/
	if(key) 
	{
		jarray = json_object_object_get(jobj, key); /*Getting the array if it is a key value pair*/
	}

#ifdef CUDA_OBS 
	printf("%s : key %s \n", __func__, key ) ;
#endif 

	int arraylen = json_object_array_length(jarray); /*Getting the length of the array*/
	// printf("Array Length: %d\n",arraylen);
	int i;
	json_object * jvalue;

	if (!( strcmp ( "interval_factor", key )))
	{
		if ( arraylen != CUBE_INFO_CNT )
		{
			printf("%s : missing element in \"interval\"\n", __func__ ) ;
			return ;
		}
		for (i=0; i< arraylen; i++)
		{
			jvalue = json_object_array_get_idx(jarray, i); /*Getting the array element at position i*/

			csp->interval_factor[i] = json_object_get_int( jvalue ) ;
		}

	} else
	{
		for (i=0; i< arraylen; i++)
		{
			jvalue = json_object_array_get_idx(jarray, i); /*Getting the array element at position i*/
			type = json_object_get_type(jvalue);
			if (type == json_type_array) {
				json_parse_array( csp, jvalue, NULL);
			}
			else if (type != json_type_object) {
				printf("value[%d]: ",i);
				print_json_value(jvalue);
			}
			else {
				json_parse( csp, jvalue);
			}
		}
	}
}

/*Parsing the json object*/
int
json_parse( struct cs_config *csp, json_object * jobj) 
{
	int i ;
	enum json_type type;
	json_object_object_foreach(jobj, key, val) 
	{ /*Passing through every array element*/
		type = json_object_get_type(val);
		// printf("key \"%s\" -- type: %d ",key, type);
		switch (type) {
		case json_type_boolean: 
		case json_type_double: 
			print_json_value(val);
			break ;

		case json_type_int: 
			i = json_object_get_int( val ) ;
			cs_config_getvalue ( csp, key, ( void *)&i) ;
			// print_json_value(val);
			break; 

		case json_type_string:
			cs_config_getvalue ( csp, key, ( void *)json_object_get_string( val )) ;
			// print_json_value(val);
			break; 
		case json_type_object:
			printf("json_type_object\n");
			jobj = json_object_object_get(jobj, key);
			json_parse( csp, jobj); 
			break;
		case json_type_array: 
			// printf("type: json_type_array, ");
			json_parse_array( csp, jobj, key);
			break;
		default:
		  assert(0); // json_type_null or any other value should not happen
		}
	}
	return ( 1 ) ;
} 

void 
cs_config_p( struct cs_config *csp )
{
	printf("cs_config_p: ------------------------------------------------------%p\n", csp ) ;
	
	printf("adj x %d y %d dbg %x do_perm %d perd %s\n",
		csp->adj_x,
		csp->adj_y,
		csp->dbg_flag,
		csp->do_permutation,
		csp->permdir );

	printf("video_src %s \n", csp->video_src ) ;

	// printf("finname %s\n", csp->finname ) ;

	printf("comp_ratio %d md %d %d %d cube %d %d %d do_cube %d\n",
		csp->comp_ratio,
		csp->md_x,
		csp->md_y,
		csp->md_z,
		csp->cubex,
		csp->cubey,
		csp->cubez, 
		csp->do_cube ) ;

	printf("frame exp %d %d %d edge %d %d display %d %d \n",
		csp->xadd,
		csp->yadd,
		csp->zadd,
		csp->edge_x,
		csp->edge_y,
		csp->disp_th_x,
		csp->disp_th_y ) ;

	// -d : frame size
	printf("frame size %d %d no seek %d md out %s\n",
		csp->frame_x,
		csp->frame_y,
		csp->do_not_seek,
		csp->md_outputfile ) ;

	printf("output file %s\n", csp->foutname ) ;

	printf("do one %d swap %d yonly %d overlap %d %d %d block %d %d %d\n",
		csp->do_one,
		csp->do_swap,
		csp->y_only,
		csp->overlap_x,
		csp->overlap_y,
		csp->overlap_z,
		csp->x_block,
		csp->y_block,
		csp->z_block ) ;

	printf("weight %d ana %d block %d comp %d interpo %d display_threshold %f\n",
		csp->weight_scheme,
		csp->do_analysis,
		csp->do_block,
		csp->do_comp_ratio,
		csp->do_interpolate,
	   	csp->display_threshold ) ;

	printf("reconstruction %d fps %d do_display %d ignore_edge %d capture %d\n",
		csp->do_reconstruction,
		csp->fps,
		csp->do_display,
		csp->ignore_edge,
	   	csp->capture ) ;

	printf("do_analysis %d interval %d %d %d \n",
		csp->do_analysis,
		csp->interval_factor[0],
		csp->interval_factor[1],
		csp->interval_factor[2] ) ;

	printf("cs_config_p: ------------------------------------------------------\n" ) ;
}

void 
cs_config_init( struct cs_config *csp )
{
	
	csp->adj_x = 0 ;
	csp->adj_y = 0 ;

	// -f : debug flag ;
	csp->dbg_flag = 0 ;

	// -p : do permutation 
	csp->do_permutation = 0 ;
	csp->permdir[0] = 0 ;

	// -I : ip cam string
	csp->video_src[0] = 0 ;
	csp->video_source = 0 ;

	// -i : input data file
	// csp->finname[0] = 0 ;

	csp->fps = 0 ;
	csp->do_display = 0 ;

	// -z : comp_ratio
	csp->comp_ratio = 0 ;

	// ignore edge: 1: yes ... i.e. no arrow for motion detection
	csp->ignore_edge = 0 ;

	// -m : motion detection
	csp->md_x = -1 ;
	csp->md_y = -1 ;
	csp->md_z = -1 ;

	// -c : cube
	csp->cubex = 0 ;
	csp->cubey = 0 ;
	csp->cubez = 0 ;

	csp->do_cube = 0 ; // derived

	// -e : frame expansion
	csp->xadd = 0 ;
	csp->yadd = 0 ;
	csp->zadd = 0 ;

	// -g : edge
	csp->edge_x = -1 ;
	csp->edge_y = -1 ;

	// -T : display threshold 
	csp->disp_th_x = 0 ;
	csp->disp_th_y = 0 ;

	// -d : frame size
	csp->frame_x = -1 ;
	csp->frame_y = -10 ;

	// -n : do not seek
	csp->do_not_seek = 0 ;

	// -F : md output file
	csp->md_outputfile[0] = 0 ;

	// -o : output file name
	csp->foutname[0] = 0 ;

	// -q
	csp->do_one = 0x7fffffff ;

	// -s
	csp->do_swap = 0 ;

	// -y
	csp->y_only = 0 ;

	// -O : overlap
	csp->overlap_x = 0 ;
	csp->overlap_y = 0 ;
	csp->overlap_z = 0 ;

	// -w : weight scheme
	csp->weight_scheme = NO_WEIGHT ;

	// -b : block size
	csp->x_block = -1 ;
	csp->y_block = -1 ;
	csp->z_block = -1 ;

	// reconstruction ;
	csp->do_reconstruction = 0.0 ;

	// capture
	csp->capture = 0 ;

	// display threshold
	csp->display_threshold = 0 ;

	// quantization

	csp->interval_factor[0] = 200 ;
	csp->interval_factor[1] = 200 ;
	csp->interval_factor[2] = 200 ;

	// global
	
	csp->do_analysis = 0 ;
	csp->do_block = 0	;
	csp->do_comp_ratio = 0 ;
	
	csp->do_interpolate = 1 ;
}

int
cs_config( char *jsonfile, struct cs_config *csp ) 
{
	int ret ;

	printf("cs_config: JSON string: %s\n", jsonfile ) ;
	json_object *jobj = json_object_from_file( jsonfile ) ;     

	ret = json_parse( csp, jobj);

	return ( ret ) ;
}

void
cs_config_getvalue( struct cs_config *csp, char *key, void *val )
{
	int i = *( int *)val ;
	char *charp = ( char *)val ;

	if ( !strcmp ( key, "adj_x" ))
	{
		csp->adj_x = i ;
	} else if ( !strcmp ( key, "adj_y" ))
	{
		csp->adj_y = i ;
	} else if ( !strcmp ( key, "dbg_flag" ))
	{
		if (( charp[0] == '0' ) && ( charp[1] == 'x' ))
			i = strtol ( charp, NULL, 16 ) ;
		else
			i = 0 ;
		csp->dbg_flag = i ;
	} else if ( !strcmp ( key, "do_display" ))
	{
		csp->do_display = i ;
	} else if ( !strcmp ( key, "fps" ))
	{
		csp->fps = i ;
	} else if ( !strcmp ( key, "do_permutation" ))
	{
		csp->do_permutation = i ;
	} else if ( !strcmp ( key, "comp_ratio" ))
	{
		csp->comp_ratio = i ;
	} else if ( !strcmp ( key, "md_x" ))
	{
		csp->md_x = i ;
	} else if ( !strcmp ( key, "md_y" ))
	{
		csp->md_y = i ;
	} else if ( !strcmp ( key, "md_z" ))
	{
		csp->md_z = i ;
	} else if ( !strcmp ( key, "doanalysis" ))
	{
		csp->do_analysis = i ;
	} else if ( !strcmp ( key, "cubex" ))
	{
		csp->do_cube++ ;
		csp->cubex = i ;
	} else if ( !strcmp ( key, "cubey" ))
	{
		csp->do_cube++ ;
		csp->cubey = i ;
	} else if ( !strcmp ( key, "cubez" ))
	{
		csp->do_cube++ ;
		csp->cubez = i ;
	} else if ( !strcmp ( key, "xadd" ))
	{
		csp->xadd = i ;
	} else if ( !strcmp ( key, "yadd" ))
	{
		csp->yadd = i ;
	} else if ( !strcmp ( key, "zadd" ))
	{
		csp->zadd = i ;
	} else if ( !strcmp ( key, "edge_x" ))
	{
		csp->edge_x = i ;
	} else if ( !strcmp ( key, "edge_y" ))
	{
		csp->edge_y = i ;
	} else if ( !strcmp ( key, "ignore_edge" ))
	{
		csp->ignore_edge = i ;
	} else if ( !strcmp ( key, "disp_th_x" ))
	{
		csp->disp_th_x = i ;
	} else if ( !strcmp ( key, "disp_th_y" ))
	{
		csp->disp_th_y = i ;
	} else if ( !strcmp ( key, "frame_x" ))
	{
		csp->frame_x = i ;
	} else if ( !strcmp ( key, "frame_y" ))
	{
		csp->frame_y = i ;
	} else if ( !strcmp ( key, "do_not_seek" ))
	{
		csp->do_not_seek = i ;
	} else if ( !strcmp ( key, "display_threshold" ))
	{
		csp->display_threshold = ( float )i / 100 ;
	} else if ( !strcmp ( key, "do_one" ))
	{
		csp->do_one = i ;
	} else if ( !strcmp ( key, "do_swap" ))
	{
		csp->do_swap = i ;
	} else if ( !strcmp ( key, "y_only" ))
	{
		csp->y_only = i ;
	} else if ( !strcmp ( key, "overlap_x" ))
	{
		csp->do_block++ ;
		csp->overlap_x = i ;
	} else if ( !strcmp ( key, "overlap_y" ))
	{
		csp->do_block++ ;
		csp->overlap_y = i ;
	} else if ( !strcmp ( key, "overlap_z" ))
	{
		csp->do_block++ ;
		csp->overlap_z = i ;
	} else if ( !strcmp ( key, "weight_scheme" ))
	{
		csp->do_block++ ;
		csp->weight_scheme = i ;
	} else if ( !strcmp ( key, "x_block" ))
	{
		csp->do_block++ ;
		csp->x_block = i ;
	} else if ( !strcmp ( key, "y_block" ))
	{
		csp->do_block++ ;
		csp->y_block = i ;
	} else if ( !strcmp ( key, "z_block" ))
	{
		csp->do_block++ ;
		csp->z_block = i ;
	} else if ( !strcmp ( key, "capture"))
	{
		csp->capture = i ;
	} else if ( !strcmp ( key, "reconstruction"))
	{
		csp->do_reconstruction = i ;
	} else if ( !strcmp ( key, "video_source"))
	{
		csp->video_source = i ;
	// string ...

	} else if ( !strcmp ( key, "video_src" ))
	{
		i = strlen ( charp ) ;
		if ( i < PATH_LENG )
			strcpy ( csp->video_src, charp ) ;
		else
			printf("video_src too long %d\n", i ) ;

	} else if ( !strcmp ( key, "permdir" ))
	{
		i = strlen ( charp ) ;
		if ( i < PATH_LENG )
			strcpy ( csp->permdir, charp ) ;
		else
			printf("permdir too long %d\n", i ) ;

#ifdef CUDA_OBS 
	} else if ( !strcmp ( key, "finname" ))
	{
		i = strlen ( charp ) ;
		if ( i < PATH_LENG )
			strcpy ( csp->finname, charp ) ;
		else
			printf("finname too long %d\n", i ) ;
#endif 

	} else if ( !strcmp ( key, "md_outputfile" ))
	{
		i = strlen ( charp ) ;
		if ( i < PATH_LENG )
			strcpy ( csp->md_outputfile, charp ) ;
		else
			printf("permdir too long %d\n", i ) ;

	} else if ( !strcmp ( key, "foutname" ))
	{
		i = strlen ( charp ) ;
		if ( i < PATH_LENG )
			strcpy ( csp->foutname, charp ) ;
		else
			printf("foutname too long %d\n", i ) ;

	} else
	{
		printf("wrong key %s \n", key ) ;
	}
}
